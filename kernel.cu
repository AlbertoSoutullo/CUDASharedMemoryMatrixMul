#include "hip/hip_runtime.h"
// MP 2: Due Sunday, Dec 16, 2012 at 11:59 p.m. PST
#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <stdio.h>
#include <windows.h>

#define TILE_WIDTH 16

__host__  void getError()
{
	hipError_t error = hipGetLastError();
	if (hipSuccess != error)
		printf("%s\n", hipGetErrorString(error));
	fflush(stdout);
}

////////////////////////////METODOS-MATRICES/////////////////////////////////////////////////////////

__host__ int readMatrix(char* fileName, float** m1, int transpose)
{
	FILE* f1 = NULL;
	int rowNum = 0;
	int colNum = 0;

	f1 = fopen(fileName, "r");

	if (f1 == NULL)
	{
		printf("The file couldn't be open.\n");
		return 0;
	}
	else
	{
		fread(&(rowNum), sizeof(int), 1, f1);
		fread(&(colNum), sizeof(int), 1, f1);
		*m1 = (float*)malloc(sizeof(float)*colNum*rowNum);
		if (transpose)
		{
			int i = 0;
			for (i = 0; i < colNum; i++)
			{
				int j = 0;
				for (j = 0; j < rowNum; j++)
				{
					fread(&(*m1)[j*rowNum + i], sizeof(float), 1, f1);
				}
			}
		}
		else
		{

			int i = 0;
			for (i = 0; i < rowNum; i++)
			{
				int j = 0;
				for (j = 0; j < rowNum; j++)
				{
					fread(&(*m1)[i*colNum + j], sizeof(float), 1, f1);
				}

			}
		}
		fclose(f1);
		return rowNum;
	}
}

__host__ void writeMatrix(char* fileName, float* m1, int colNum, int rowNum, int transpose)
{
	FILE* f1 = NULL;

	f1 = fopen(fileName, "w");

	if (f1 == NULL)
	{
		printf("The file couldn't be open.\n");
	}
	else
	{
		fwrite(&(rowNum), sizeof(int), 1, f1);
		fwrite(&(colNum), sizeof(int), 1, f1);

		if (transpose)
		{
			int i = 0;
			for (i = 0; i < rowNum; i++)
			{
				int j = 0;
				for (j = 0; j <rowNum; j++)
				{
					fwrite(&(m1[j*rowNum + i]), sizeof(float), 1, f1);
				}
			}
		}
		else
		{
			int i = 0;
			for (i = 0; i < rowNum; i++)
			{
				int j = 0;
				for (j = 0; j <rowNum; j++)
				{
					fwrite(&(m1[i*colNum + j]), sizeof(float), 1, f1);
				}
			}
		}
		fclose(f1);
	}
}


__host__ void printMatrix(float* matrix, int rowNum, int colNum)
{
	int i = 0;
	for (i = 0; i < rowNum; i++)
	{
		int j = 0;
		for (j = 0; j < colNum; j++)
		{
			printf("%0.3f ", matrix[i*colNum + j]);
		}
		printf("\n");
	}
}

__host__ int martrixComparator(float* matrix1, float* correctMatrix, int rowNum, int colNum)
{
	int  i = 0;
	for (i = 0; i < rowNum; i++)
	{
		int  j = 0;
		for (j = 0; j < colNum; j++)
		{
			if (matrix1[i*colNum + j] != correctMatrix[i*colNum + j])
			{
				return 0;
			}
		}
	}
	return 1;
}

__host__ void multiplySimple(float* m1, float* m2, float* mres, int numFilas1, int numFilas2, int numColumnas)
{

	for (int i = 0; i < numFilas1; i++) {//iterate through a given set of rows of [A]
		for (int j = 0; j < numFilas2; j++) {//iterate through columns of [B]
			for (int k = 0; k < numColumnas; k++) {//iterate through rows of [B]
				mres[i*numColumnas + j] += (m1[i*numColumnas + k] * m2[j*numColumnas + k]);
			}
		}
	}

}


// Compute C = A * B
__global__ void matrixMultiply(float * A, float * B, float * C,
	int numARows, int numAColumns,
	int numBRows, int numBColumns,
	int numCRows, int numCColumns) {
	
	__shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
	__shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x, by = blockIdx.y,
		tx = threadIdx.x, ty = threadIdx.y,
		Row = by * TILE_WIDTH + ty,
		Col = bx * TILE_WIDTH + tx;
	float Pvalue = 0;

	for (int m = 0; m < (numAColumns - 1) / TILE_WIDTH + 1; ++m) {
		if (Row < numARows && m*TILE_WIDTH + tx < numAColumns)
			ds_M[ty][tx] = A[Row*numAColumns + m * TILE_WIDTH + tx];
		else
			ds_M[ty][tx] = 0;
		if (Col < numBColumns && m*TILE_WIDTH + ty < numBRows)
			ds_N[ty][tx] = B[(m*TILE_WIDTH + ty)*numBColumns + Col];
		else
			ds_N[ty][tx] = 0;

		__syncthreads();
		for (int k = 0; k < TILE_WIDTH; ++k)
			Pvalue += ds_M[ty][k] * ds_N[k][tx];
		__syncthreads();
	}
	if (Row < numCRows && Col < numCColumns)
		C[Row*numCColumns + Col] = Pvalue;
}

__global__ void MatMul(float* A, float* B, float* C, int ARows, int ACols, int BRows,
	int BCols, int CRows, int CCols)
{
	float CValue = 0;

	int Row = blockIdx.y*TILE_WIDTH + threadIdx.y;
	int Col = blockIdx.x*TILE_WIDTH + threadIdx.x;

	__shared__ float As[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

	for (int k = 0; k < (TILE_WIDTH + ACols - 1) / TILE_WIDTH; k++) {

		if (k*TILE_WIDTH + threadIdx.x < ACols && Row < ARows)
			As[threadIdx.y][threadIdx.x] = A[Row*ACols + k * TILE_WIDTH + threadIdx.x];
		else
			As[threadIdx.y][threadIdx.x] = 0.0;

		if (k*TILE_WIDTH + threadIdx.y < BRows && Col < BCols)
			Bs[threadIdx.y][threadIdx.x] = B[(k*TILE_WIDTH + threadIdx.y)*BCols + Col];
		else
			Bs[threadIdx.y][threadIdx.x] = 0.0;

		__syncthreads();

		for (int n = 0; n < TILE_WIDTH; ++n)
			CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

		__syncthreads();
	}

	if (Row < CRows && Col < CCols)
		C[((blockIdx.y * blockDim.y + threadIdx.y)*CCols) +
		(blockIdx.x * blockDim.x) + threadIdx.x] = CValue;
}

int main(int argc, char ** argv) {

	//Initialazing variables
	float* h_mat1 = NULL; // The A matrix
	float* h_mat2 = NULL; // The B matrix
	float* h_matres = NULL; // The output C matrix
	float* d_mat1 = NULL;
	float* d_mat2 = NULL;
	float* d_matres = NULL;


	//File names
	/*char str1[20];
	char str2[20];
	printf("Introduce the fila name of the first matrix:");
	scanf("%s", str1);
	printf("Introduce the fila name of the second matrix:");
	scanf("%s", str2);*/
	clock_t cpu_startTimeTotal, cpu_endTimeTotal, cpu_startTimeMult, cpu_endTimeMult;
	double cpu_ElapseTimeTotal = 0;
	double cpu_ElapseTimeMult = 0;
	cpu_startTimeTotal = clock();


	int numARows; // number of rows in the matrix A
	int numAColumns; // number of columns in the matrix A
	numAColumns = numARows = readMatrix("500x500.bin", &h_mat1, 0);

	int numBRows; // number of rows in the matrix B
	int numBColumns; // number of columns in the matrix B
	numBColumns = numBRows = readMatrix("500x500I.bin", &h_mat2, 1);


	//inicializar resultado 
	h_matres = (float*)malloc(sizeof(float)*numAColumns*numBRows);


	//Reservamos memoria en GPU
	hipMalloc(&d_mat1, sizeof(float) * numARows * numAColumns);
	hipMalloc(&d_mat2, sizeof(float) * numBRows * numBColumns);
	hipMalloc(&d_matres, sizeof(float) * numAColumns * numBRows);



	//pasamos las matrices a memoria de GPU
	hipMemcpy(d_mat1, h_mat1, sizeof(float) * numARows * numAColumns, hipMemcpyHostToDevice);
	getError();
	hipMemcpy(d_mat2, h_mat2, sizeof(float) * numBRows * numBColumns, hipMemcpyHostToDevice);
	getError();



	//Definimos el tama�o de bloque 
	int tam = numARows * numAColumns;
	int numthreadporbloque = 1024;

	int numbloques = ((numARows / 32) * (numBRows / 32)) + 1;
	int numbloquesMax = 500;
	int totalDivision = numbloques / numbloquesMax + 1;

	cpu_startTimeMult = clock();

	
	//Multiplicamos
	MatMul << <numbloquesMax, numthreadporbloque >> >(d_mat1, d_mat2, d_matres,
		numARows, numAColumns,
		numBRows, numBColumns,
		numAColumns, numBRows);

	hipDeviceSynchronize();
	getError();


	cpu_endTimeMult = clock();


	//Pasamos la matriz resultado a cpu
	hipMemcpy(h_matres, d_matres, sizeof(float) * numAColumns * numBRows, hipMemcpyDeviceToHost);
	getError();

	hipFree(d_mat1);
	hipFree(d_mat2);
	hipFree(d_matres);

	//escribimos la matriz resultado en un archivo
	writeMatrix("result.bin", h_matres, numAColumns, numBRows, 0);

	cpu_endTimeTotal = clock();
	cpu_ElapseTimeMult = ((cpu_endTimeMult - cpu_startTimeMult) / CLOCKS_PER_SEC);
	cpu_ElapseTimeTotal = ((cpu_endTimeTotal - cpu_startTimeTotal) / CLOCKS_PER_SEC);

	printf("Tiempo Total: %d", cpu_ElapseTimeTotal);
	printf("Tiempo Multiplicando: %d", cpu_ElapseTimeMult);

	//int result = martrixComparator(h_mat1, h_matres, numAColumns, numBRows);

	//if (result) printf("Matrix match.\n");

	free(h_mat1);
	free(h_mat2);
	free(h_matres);

	

	return 0;
}